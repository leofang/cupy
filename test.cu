#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_runtime.h>
#define checkCudaErrors(x) {assert (0==x);}


int main() {
    int nGPUs = 2;
    int whichGPUs[] = {0, 1};

    // start and end from device 0
    // when we start a fresh program, the peer access is disabled by default
    // (we check the peer access status by trial and error)
    printf("before creating a plan...\n");
    checkCudaErrors(hipSetDevice(0));
    for (int i=0; i<2; i++) {
        hipSetDevice(i);
	hipError_t status = hipDeviceDisablePeerAccess((i+1)%2);
        if (status != hipErrorPeerAccessNotEnabled) {
            printf("dev: %i, status: %s\n", i, hipGetErrorString(status));
	    exit(-1);
        }
    }
    checkCudaErrors(hipSetDevice(0));

    // cufftCreate() - Create an empty plan
    hipfftResult result;
    hipfftHandle plan_input;
    checkCudaErrors (hipfftCreate (&plan_input));

    // cufftXtSetGPUs() - Define which GPUs to use
    result = hipfftXtSetGPUs (plan_input, nGPUs, whichGPUs);
    if (result != HIPFFT_SUCCESS)
    {
        printf ("hipfftXtSetGPUs failed\n"); exit (EXIT_FAILURE);
	exit(-1);
    }

    int new_size = 128;
    size_t* worksize;
    worksize =(size_t*)malloc(sizeof(size_t) * nGPUs);

    // cufftMakePlan1d() - Create the plan
    checkCudaErrors(hipfftMakePlan1d(plan_input, new_size, HIPFFT_C2C, 1, worksize));

    // start and end from device 0
    // cuFFT internally enabled bidirectional access upon this point
    printf("after creating a plan/before destroying it...\n");
    checkCudaErrors(hipSetDevice(0));
    for (int i=0; i<2; i++) {
        hipSetDevice(i);
	hipError_t status = hipDeviceEnablePeerAccess((i+1)%2, 0);
        if (status != hipErrorPeerAccessAlreadyEnabled) {
            printf("dev: %i, status: %s\n", i, hipGetErrorString(status));
	    exit(-1);
        }
    }
    checkCudaErrors(hipSetDevice(0));

    // cufftDestroy() - Destroy FFT plan
    checkCudaErrors(hipfftDestroy(plan_input));

    // start and end from device 0
    // cuFFT only disabled 1->0 access, but not 0->1 access, so we error out at i=1
    printf("after destroying the plan...\n");
    checkCudaErrors(hipSetDevice(0));
    for (int i=0; i<2; i++) {
        hipSetDevice(i);
	hipError_t status = hipDeviceDisablePeerAccess((i+1)%2);
        if (status != hipErrorPeerAccessNotEnabled) {
            printf("dev: %i, status: %s\n", i, hipGetErrorString(status));
	    exit(-1);
        }
    }
    checkCudaErrors(hipSetDevice(0));

    free(worksize);

    return 0;
}
