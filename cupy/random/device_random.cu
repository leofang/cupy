#include "device_random.h"

#ifdef CUPY_USE_HIP

#include <hip/hip_runtime.h>
#include <hiprand_kernel.h>
typedef hiprandState hiprandState;
typedef hiprandStateMRG32k3a hiprandStateMRG32k3a;
typedef hiprandStatePhilox4_32_10_t hiprandStatePhilox4_32_10_t;

#elif !defined(CUPY_NO_CUDA)

#include <hiprand/hiprand_kernel.h>

#endif

size_t get_curandState_size() {
    return sizeof(hiprandState);
}

size_t get_curandStateMRG32k3a_size() {
    return sizeof(hiprandStateMRG32k3a);
}

size_t get_curandStatePhilox4_32_10_t_size() {
    return sizeof(hiprandStatePhilox4_32_10_t);
}
