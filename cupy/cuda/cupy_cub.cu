#include "hip/hip_runtime.h"
#include "cupy_cub.h"  // need to make atomicAdd visible to CUB templates early
#include <cupy/type_dispatcher.cuh>


namespace cupy {



}  // namespace cupy



//
// APIs exposed to CuPy
//

/* -------- device reduce -------- */

void cub_device_reduce(void* workspace, size_t& workspace_size, void* x, void* y,
    int num_items, hipStream_t stream, int op, int dtype_id)
{
    switch(op) {
    case CUPY_CUB_SUM:      return dtype_dispatcher(dtype_id, _cub_reduce_sum(),
                                workspace, workspace_size, x, y, num_items, stream);
    case CUPY_CUB_MIN:      return dtype_dispatcher(dtype_id, _cub_reduce_min(),
                                workspace, workspace_size, x, y, num_items, stream);
    case CUPY_CUB_MAX:      return dtype_dispatcher(dtype_id, _cub_reduce_max(),
                                workspace, workspace_size, x, y, num_items, stream);
    case CUPY_CUB_ARGMIN:   return dtype_dispatcher(dtype_id, _cub_reduce_argmin(),
                                workspace, workspace_size, x, y, num_items, stream);
    case CUPY_CUB_ARGMAX:   return dtype_dispatcher(dtype_id, _cub_reduce_argmax(),
                                workspace, workspace_size, x, y, num_items, stream);
    case CUPY_CUB_PROD:     return dtype_dispatcher(dtype_id, _cub_reduce_prod(),
                                workspace, workspace_size, x, y, num_items, stream);
    default:            throw std::runtime_error("Unsupported operation");
    }
}

size_t cub_device_reduce_get_workspace_size(void* x, void* y, int num_items,
    hipStream_t stream, int op, int dtype_id)
{
    size_t workspace_size = 0;
    cub_device_reduce(NULL, workspace_size, x, y, num_items, stream,
                      op, dtype_id);
    return workspace_size;
}

/* -------- device segmented reduce -------- */

void cub_device_segmented_reduce(void* workspace, size_t& workspace_size,
    void* x, void* y, int num_segments, int segment_size,
    hipStream_t stream, int op, int dtype_id)
{
    // CUB internally use int for offset...
    // This iterates over [0, segment_size, 2*segment_size, 3*segment_size, ...]
    #ifndef CUPY_USE_HIP
    CountingInputIterator<int> count_itr(0);
    #else
    rocprim::counting_iterator<int> count_itr(0);
    #endif
    _arange scaling(segment_size);
    seg_offset_itr itr(count_itr, scaling);

    switch(op) {
    case CUPY_CUB_SUM:
        return dtype_dispatcher(dtype_id, _cub_segmented_reduce_sum(),
                   workspace, workspace_size, x, y, num_segments, itr, stream);
    case CUPY_CUB_MIN:
        return dtype_dispatcher(dtype_id, _cub_segmented_reduce_min(),
                   workspace, workspace_size, x, y, num_segments, itr, stream);
    case CUPY_CUB_MAX:
        return dtype_dispatcher(dtype_id, _cub_segmented_reduce_max(),
                   workspace, workspace_size, x, y, num_segments, itr, stream);
    case CUPY_CUB_PROD:
        return dtype_dispatcher(dtype_id, _cub_segmented_reduce_prod(),
                   workspace, workspace_size, x, y, num_segments, itr, stream);
    default:
        throw std::runtime_error("Unsupported operation");
    }
}

size_t cub_device_segmented_reduce_get_workspace_size(void* x, void* y,
    int num_segments, int segment_size,
    hipStream_t stream, int op, int dtype_id)
{
    size_t workspace_size = 0;
    cub_device_segmented_reduce(NULL, workspace_size, x, y,
                                num_segments, segment_size, stream,
                                op, dtype_id);
    return workspace_size;
}

/*--------- device spmv (sparse-matrix dense-vector multiply) ---------*/

void cub_device_spmv(void* workspace, size_t& workspace_size, void* values,
    void* row_offsets, void* column_indices, void* x, void* y, int num_rows,
    int num_cols, int num_nonzeros, hipStream_t stream,
    int dtype_id)
{
    #ifndef CUPY_USE_HIP
    return dtype_dispatcher(dtype_id, _cub_device_spmv(),
                            workspace, workspace_size, values, row_offsets,
                            column_indices, x, y, num_rows, num_cols,
                            num_nonzeros, stream);
    #endif
}

size_t cub_device_spmv_get_workspace_size(void* values, void* row_offsets,
    void* column_indices, void* x, void* y, int num_rows, int num_cols,
    int num_nonzeros, hipStream_t stream, int dtype_id)
{
    size_t workspace_size = 0;
    #ifndef CUPY_USE_HIP
    cub_device_spmv(NULL, workspace_size, values, row_offsets, column_indices,
                    x, y, num_rows, num_cols, num_nonzeros, stream, dtype_id);
    #endif
    return workspace_size;
}

/* -------- device scan -------- */

void cub_device_scan(void* workspace, size_t& workspace_size, void* x, void* y,
    int num_items, hipStream_t stream, int op, int dtype_id)
{
    switch(op) {
    case CUPY_CUB_CUMSUM:
        return dtype_dispatcher(dtype_id, _cub_inclusive_sum(),
                                workspace, workspace_size, x, y, num_items, stream);
    case CUPY_CUB_CUMPROD:
        return dtype_dispatcher(dtype_id, _cub_inclusive_product(),
                                workspace, workspace_size, x, y, num_items, stream);
    default:
        throw std::runtime_error("Unsupported operation");
    }
}

size_t cub_device_scan_get_workspace_size(void* x, void* y, int num_items,
    hipStream_t stream, int op, int dtype_id)
{
    size_t workspace_size = 0;
    cub_device_scan(NULL, workspace_size, x, y, num_items, stream,
                    op, dtype_id);
    return workspace_size;
}

/* -------- device histogram -------- */

void cub_device_histogram_range(void* workspace, size_t& workspace_size, void* x, void* y,
    int n_bins, void* bins, size_t n_samples, hipStream_t stream, int dtype_id)
{
    // TODO(leofang): support complex
    if (dtype_id == CUPY_TYPE_COMPLEX64 || dtype_id == CUPY_TYPE_COMPLEX128) {
	    throw std::runtime_error("complex dtype is not yet supported");
    }

    // TODO(leofang): n_samples is of type size_t, but if it's < 2^31 we cast it to int later
    return dtype_dispatcher(dtype_id, _cub_histogram_range(),
                            workspace, workspace_size, x, y, n_bins, bins, n_samples, stream);
}

size_t cub_device_histogram_range_get_workspace_size(void* x, void* y, int n_bins,
    void* bins, size_t n_samples, hipStream_t stream, int dtype_id)
{
    size_t workspace_size = 0;
    cub_device_histogram_range(NULL, workspace_size, x, y, n_bins, bins, n_samples,
                               stream, dtype_id);
    return workspace_size;
}

void cub_device_histogram_even(void* workspace, size_t& workspace_size, void* x, void* y,
    int n_bins, int lower, int upper, size_t n_samples, hipStream_t stream, int dtype_id)
{
    #ifndef CUPY_USE_HIP
    return dtype_dispatcher(dtype_id, _cub_histogram_even(),
                            workspace, workspace_size, x, y, n_bins, lower, upper, n_samples, stream);
    #endif
}

size_t cub_device_histogram_even_get_workspace_size(void* x, void* y, int n_bins,
    int lower, int upper, size_t n_samples, hipStream_t stream, int dtype_id)
{
    size_t workspace_size = 0;
    cub_device_histogram_even(NULL, workspace_size, x, y, n_bins, lower, upper, n_samples,
                              stream, dtype_id);
    return workspace_size;
}
