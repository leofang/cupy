#include "hip/hip_runtime.h"
#include <cupy/complex.cuh>
#include <cupy/cub/device/device_reduce.cuh>
#include "cupy_cub.h"
#include <stdexcept>

using namespace hipcub;

// Minimum boilerplate to support complex numbers in sum(), min(), and max():
// - This works only because all data fields in the *Traits struct are not
//   used in <cub/device/device_reduce.cuh>.
// - DO NOT USE THIS STUB for supporting CUB sorting!!!!!!
// - The Max() and Lowest() below are chosen to comply with NumPy's lexical
//   ordering; note that std::numeric_limits<T> does not support complex
//   numbers as in general the comparison is ill defined.
template <>
struct FpLimits<complex<float>>
{
    static __host__ __device__ __forceinline__ complex<float> Max() {
        return (complex<float>(FLT_MAX, FLT_MAX));
    }

    static __host__ __device__ __forceinline__ complex<float> Lowest() {
        return (complex<float>(FLT_MAX * float(-1), FLT_MAX * float(-1)));
    }
};

template <>
struct FpLimits<complex<double>>
{
    static __host__ __device__ __forceinline__ complex<double> Max() {
        return (complex<double>(DBL_MAX, DBL_MAX));
    }

    static __host__ __device__ __forceinline__ complex<double> Lowest() {
        return (complex<double>(DBL_MAX * double(-1), DBL_MAX * double(-1)));
    }
};

template <> struct NumericTraits<complex<float>>  : BaseTraits<FLOATING_POINT, true, false, unsigned int, complex<float>> {};
template <> struct NumericTraits<complex<double>> : BaseTraits<FLOATING_POINT, true, false, unsigned long long, complex<double>> {};
// end of boilerplate


//
// **** dtype_dispatcher ****
//
// This is implemented with reference to the following implementation.
// https://github.com/rapidsai/cudf/blob/branch-0.6/cpp/src/utilities/type_dispatcher.hpp
//
template <class functor_t, typename... Ts>
void dtype_dispatcher(int dtype_id, functor_t f, Ts&&... args)
{
    switch (dtype_id) {
    case CUPY_CUB_INT8:	      return f.template operator()<char>(std::forward<Ts>(args)...);
    case CUPY_CUB_INT16:      return f.template operator()<short>(std::forward<Ts>(args)...);
    case CUPY_CUB_INT32:      return f.template operator()<int>(std::forward<Ts>(args)...);
    case CUPY_CUB_INT64:      return f.template operator()<long>(std::forward<Ts>(args)...);
    case CUPY_CUB_UINT8:      return f.template operator()<unsigned char>(std::forward<Ts>(args)...);
    case CUPY_CUB_UINT16:     return f.template operator()<unsigned short>(std::forward<Ts>(args)...);
    case CUPY_CUB_UINT32:     return f.template operator()<unsigned int>(std::forward<Ts>(args)...);
    case CUPY_CUB_UINT64:     return f.template operator()<unsigned long>(std::forward<Ts>(args)...);
    case CUPY_CUB_FLOAT32:    return f.template operator()<float>(std::forward<Ts>(args)...);
    case CUPY_CUB_FLOAT64:    return f.template operator()<double>(std::forward<Ts>(args)...);
    case CUPY_CUB_COMPLEX64:  return f.template operator()<complex<float>>(std::forward<Ts>(args)...);
    case CUPY_CUB_COMPLEX128: return f.template operator()<complex<double>>(std::forward<Ts>(args)...);
    default:
	throw std::runtime_error("Unsupported dtype ID");
    }
}

//
// **** cub_reduce_sum ****
//
struct _cub_reduce_sum {
    template <typename T>
    void operator()(void *x, void *y, int num_items, void *workspace,
        size_t &workspace_size, hipStream_t s)
    {
        DeviceReduce::Sum(workspace, workspace_size, static_cast<T*>(x),
            static_cast<T*>(y), num_items, s);
    }
};

void cub_reduce_sum(void *x, void *y, int num_items, void *workspace,
    size_t &workspace_size, hipStream_t stream, int dtype_id)
{
    dtype_dispatcher(dtype_id, _cub_reduce_sum(),
        x, y, num_items, workspace, workspace_size, stream);
}

size_t cub_reduce_sum_get_workspace_size(void *x, void *y, int num_items,
    hipStream_t stream, int dtype_id)
{
    size_t workspace_size = 0;
    cub_reduce_sum(x, y, num_items, NULL, workspace_size, stream, dtype_id);
    return workspace_size;
}

//
// **** cub_reduce_min ****
//
struct _cub_reduce_min {
    template <typename T>
    void operator()(void *x, void *y, int num_items, void *workspace,
        size_t &workspace_size, hipStream_t s)
    {
        DeviceReduce::Min(workspace, workspace_size, static_cast<T*>(x),
            static_cast<T*>(y), num_items, s);
    }
};

void cub_reduce_min(void *x, void *y, int num_items, void *workspace,
    size_t &workspace_size, hipStream_t stream, int dtype_id)
{
    dtype_dispatcher(dtype_id, _cub_reduce_min(),
        x, y, num_items, workspace, workspace_size, stream);
}

size_t cub_reduce_min_get_workspace_size(void *x, void *y, int num_items,
    hipStream_t stream, int dtype_id)
{
    size_t workspace_size = 0;
    cub_reduce_min(x, y, num_items, NULL, workspace_size, stream, dtype_id);
    return workspace_size;
}

//
// **** cub_reduce_max ****
//
struct _cub_reduce_max {
    template <typename T>
    void operator()(void *x, void *y, int num_items, void *workspace,
        size_t &workspace_size, hipStream_t s)
    {
        DeviceReduce::Max(workspace, workspace_size, static_cast<T*>(x),
            static_cast<T*>(y), num_items, s);
    }
};

void cub_reduce_max(void *x, void *y, int num_items, void *workspace,
    size_t &workspace_size, hipStream_t stream, int dtype_id)
{
    dtype_dispatcher(dtype_id, _cub_reduce_max(),
        x, y, num_items, workspace, workspace_size, stream);
}

size_t cub_reduce_max_get_workspace_size(void *x, void *y, int num_items,
    hipStream_t stream, int dtype_id)
{
    size_t workspace_size = 0;
    cub_reduce_max(x, y, num_items, NULL, workspace_size, stream, dtype_id);
    return workspace_size;
}
