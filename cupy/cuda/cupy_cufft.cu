#include "cupy_cufft.h"


hipfftResult setCallback(hipfftHandle plan, void **callbackRoutine,
                        hipfftXtCallbackType type, void **callerInfo)
{
    return hipfftXtSetCallback(plan, callbackRoutine, type, callerInfo);
}


hipfftResult clearCallback(hipfftHandle plan, hipfftXtCallbackType type)
{
    return hipfftXtClearCallback(plan, type);
}
