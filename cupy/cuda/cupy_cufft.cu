#include "cupy_cufft.h"


//extern __device__ hipfftCallbackLoadC CUPY_device_cufft_callback_load_complex64;
//extern __device__ hipfftCallbackLoadZ CUPY_device_cufft_callback_load_complex128;
//extern __device__ hipfftCallbackLoadR CUPY_device_cufft_callback_load_float32;
//extern __device__ hipfftCallbackLoadD CUPY_device_cufft_callback_load_float64;
//extern __device__ hipfftCallbackStoreC CUPY_device_cufft_callback_store_complex64;
//extern __device__ hipfftCallbackStoreZ CUPY_device_cufft_callback_store_complex128;
//extern __device__ hipfftCallbackStoreR CUPY_device_cufft_callback_store_float32;
//extern __device__ hipfftCallbackStoreD CUPY_device_cufft_callback_store_float64;

//hipfftCallbackLoadR CUPY_host_cufft_callback_load_float32;
//hipfftCallbackLoadD CUPY_host_cufft_callback_load_float64;
//hipfftCallbackStoreR CUPY_host_cufft_callback_store_float32;
//hipfftCallbackStoreD CUPY_host_cufft_callback_store_float64;


hipfftResult setCallbackLoadC(hipfftHandle plan, intptr_t ptr, void** callerInfo) {
    hipfftCallbackLoadC CUPY_host_cufft_callback_load_complex64 = (hipfftCallbackLoadC)ptr;
    return hipfftXtSetCallback(plan, (void**)&CUPY_host_cufft_callback_load_complex64, HIPFFT_CB_LD_COMPLEX, callerInfo);
}

hipfftResult setCallbackLoadZ(hipfftHandle plan, intptr_t ptr, void** callerInfo) {
    hipfftCallbackLoadZ CUPY_host_cufft_callback_load_complex128 = (hipfftCallbackLoadZ)ptr;
    return hipfftXtSetCallback(plan, (void**)&CUPY_host_cufft_callback_load_complex128, HIPFFT_CB_LD_COMPLEX_DOUBLE, callerInfo);
}

//hipfftResult setCallbackLoadR(hipfftHandle plan, void** callerInfo) {
//    return hipfftXtSetCallback(plan, (void**)&CUPY_host_cufft_callback_load_float32, HIPFFT_CB_LD_REAL, callerInfo);
//}
//
//hipfftResult setCallbackLoadD(hipfftHandle plan, void** callerInfo) {
//    return hipfftXtSetCallback(plan, (void**)&CUPY_host_cufft_callback_load_float64, HIPFFT_CB_LD_REAL_DOUBLE, callerInfo);
//}

hipfftResult setCallbackStoreC(hipfftHandle plan, intptr_t ptr, void** callerInfo) {
    hipfftCallbackStoreC CUPY_host_cufft_callback_store_complex64 = (hipfftCallbackStoreC)ptr;
    return hipfftXtSetCallback(plan, (void**)&CUPY_host_cufft_callback_store_complex64, HIPFFT_CB_ST_COMPLEX, callerInfo);
}

hipfftResult setCallbackStoreZ(hipfftHandle plan, intptr_t ptr, void** callerInfo) {
    hipfftCallbackStoreZ CUPY_host_cufft_callback_store_complex128 = (hipfftCallbackStoreZ)ptr;
    return hipfftXtSetCallback(plan, (void**)&CUPY_host_cufft_callback_store_complex128, HIPFFT_CB_ST_COMPLEX_DOUBLE, callerInfo);
}

//hipfftResult setCallbackStoreR(hipfftHandle plan, void** callerInfo) {
//    return hipfftXtSetCallback(plan, (void**)&CUPY_host_cufft_callback_store_float32, HIPFFT_CB_ST_REAL, callerInfo);
//}
//
//hipfftResult setCallbackStoreD(hipfftHandle plan, void** callerInfo) {
//    return hipfftXtSetCallback(plan, (void**)&CUPY_host_cufft_callback_store_float64, HIPFFT_CB_ST_REAL_DOUBLE, callerInfo);
//}
